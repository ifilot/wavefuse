#include "hip/hip_runtime.h"
 /*************************************************************************
 *   This file is part of Wavefuse                                        *
 *   https://gitlab.tue.nl/ifilot/Wavefuse                                *
 *                                                                        *
 *   Author: Ivo Filot <i.a.w.filot@tue.nl>                               *
 *                                                                        *
 *   Wavefuse is free software: you can redistribute it and/or modify     *
 *   it under the terms of the GNU General Public License as published    *
 *   by the Free Software Foundation, either version 3 of the License,    *
 *   or (at your option) any later version.                               *
 *                                                                        *
 *   Wavefuse is distributed in the hope that it will be useful,          *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty          *
 *   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.              *
 *   See the GNU General Public License for more details.                 *
 *                                                                        *
 *   You should have received a copy of the GNU General Public License    *
 *   along with this program.  If not, see http://www.gnu.org/licenses/.  *
 *                                                                        *
 **************************************************************************/

// add auxiliary cuda functions
#include "check_cuda.h"
#include "cuda_events.h"

// general configuration file
#include "config.h"

// include kernels
#include "kernels/laplacians.h"
#include "kernels/reaction_kinetics.h"
#include "kernels/update.h"

// other includes
#include "rd3d.h"

RD3D::RD3D() {

}

/**
 * @brief      Run time-integration on GPU
 */
void RD3D::run_cuda() {
    this->initialize_variables();

    std::cout << "Starting time-integration" << std::endl;

    std::cout << "Setting grids and blocks...           ";
    dim3 gridx(this->my / this->pencils, this->mz, 1);
    dim3 blockx(this->mx, this->pencils, 1);
    dim3 gridy(this->mx / this->pencils, this->mz, 1);
    dim3 blocky(this->pencils, this->my, 1);
    dim3 gridz(this->mx / this->pencils, this->my, 1);
    dim3 blockz(this->pencils, this->mz, 1);
    unsigned int block = this->mx;;
    unsigned int grid = (this->ncells + this->mx - 1) / this->mx;

    unsigned shared_mem_size = 0;
    if(this->zeroflux) {
        shared_mem_size = this->pencils * this->mx * sizeof(float);
    } else {
        shared_mem_size = this->pencils * (this->mx + 2) * sizeof(float);
    }
    std::cout << donestring << std::endl << std::endl;

    // keep track of time
    hipEvent_t startEvent, stopEvent;
    hipEvent_t startEventKernel, stopEventKernel;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    checkCuda( hipEventCreate(&startEventKernel) );
    checkCuda( hipEventCreate(&stopEventKernel) );

    for(unsigned int t=0; t<this->timesteps; t++) {
        // start timer and execute
        checkCuda( hipEventRecord(startEvent, 0) );

        // keep track of time
        float milliseconds = 0;
        float laplacian_x_times = 0;
        float laplacian_y_times = 0;
        float laplacian_z_times = 0;
        float reaction_times = 0;
        float update_times = 0;
        float laplacian_summation_times = 0;

        for(unsigned int i=0; i<this->tsteps; i++) {

            // calculate laplacian for A
            if(this->zeroflux) {
                // x2 derivative
                start_event(&startEventKernel);
                derivative_x2_zeroflux<<<gridx,blockx,shared_mem_size>>>(d_a, d_dx2);
                laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);

                // y2 derivative
                start_event(&startEventKernel);
                derivative_y2_zeroflux<<<gridy,blocky,shared_mem_size>>>(d_a, d_dy2);
                laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);

                // z2 derivative
                start_event(&startEventKernel);
                derivative_z2_zeroflux<<<gridz,blockz,shared_mem_size>>>(d_a, d_dz2);
                laplacian_z_times += stop_event(&startEventKernel, &stopEventKernel);;
            } else {
                // x2 derivative
                start_event(&startEventKernel);
                derivative_x2_pbc<<<gridx,blockx,shared_mem_size>>>(d_a, d_dx2);
                laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

                // y2 derivative
                start_event(&startEventKernel);
                derivative_y2_pbc<<<gridy,blocky,shared_mem_size>>>(d_a, d_dy2);
                laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;

                // z2 derivative
                start_event(&startEventKernel);
                derivative_z2_pbc<<<gridz,blockz,shared_mem_size>>>(d_a, d_dz2);
                laplacian_z_times += stop_event(&startEventKernel, &stopEventKernel);;
            }

            // sum all three derivative components
            start_event(&startEventKernel);
            construct_laplacian_a<<<grid,block>>>(d_da, d_dx2, d_dy2, d_dz2);
            laplacian_summation_times += stop_event(&startEventKernel, &stopEventKernel);;

            // calculate laplacian for B
            if(this->zeroflux) {
                // x2 derivative
                start_event(&startEventKernel);
                derivative_x2_zeroflux<<<gridx,blockx,shared_mem_size>>>(d_b, d_dx2);
                laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

                // y2 derivative
                start_event(&startEventKernel);
                derivative_y2_zeroflux<<<gridy,blocky,shared_mem_size>>>(d_b, d_dy2);
                laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;

                // z2 derivative
                start_event(&startEventKernel);
                derivative_z2_zeroflux<<<gridz,blockz,shared_mem_size>>>(d_b, d_dz2);
                laplacian_z_times += stop_event(&startEventKernel, &stopEventKernel);;
            } else {
                // x2 derivative
                start_event(&startEventKernel);
                derivative_x2_pbc<<<gridx,blockx,shared_mem_size>>>(d_b, d_dx2);
                laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

                // y2 derivative
                start_event(&startEventKernel);
                derivative_y2_pbc<<<gridy,blocky,shared_mem_size>>>(d_b, d_dy2);
                laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;

                // z2 derivative
                start_event(&startEventKernel);
                derivative_z2_pbc<<<gridz,blockz,shared_mem_size>>>(d_b, d_dz2);
                laplacian_z_times += stop_event(&startEventKernel, &stopEventKernel);;
            }

            // sum all three derivative components
            start_event(&startEventKernel);
            construct_laplacian_b<<<grid,block>>>(d_db, d_dx2, d_dy2, d_dz2);
            laplacian_summation_times += stop_event(&startEventKernel, &stopEventKernel);;

            // calculate reaction
            start_event(&startEventKernel);
            switch(this->reaction_type) {
                case KINETICS::GRAY_SCOTT:
                    reaction_gray_scott<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
                break;
                case KINETICS::BRUSSELATOR:
                    reaction_brusselator<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
                break;
                default:
                    throw std::runtime_error("Invalid reaction system encountered.");
            }
            reaction_times += stop_event(&startEventKernel, &stopEventKernel);;

            // update
            start_event(&startEventKernel);
            update<<<grid,block>>>(d_a, d_b, d_da, d_db, d_ra, d_rb);
            update_times += stop_event(&startEventKernel, &stopEventKernel);;;
        }

        // stop timer
        checkCuda( hipEventRecord(stopEvent, 0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&milliseconds, startEvent, stopEvent) );

        // copy results back
        int bytes = this->ncells * sizeof(float);
        checkCuda( hipMemcpy(this->a, this->d_a, bytes, hipMemcpyDeviceToHost) );
        checkCuda( hipMemcpy(this->b, this->d_b, bytes, hipMemcpyDeviceToHost) );

        auto start = std::chrono::system_clock::now();
        char buffer[50];
        sprintf(buffer, "data_%03i.bin", (t+1));
        if(this->store_conc == "A") {
            this->write_binary(std::string(buffer), a);
        } else if(this->store_conc == "B") {
            this->write_binary(std::string(buffer), b);
        } else {
            throw std::runtime_error("Unclear which concentration to store: " + this->store_conc);
        }

        auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;

        float other_times = milliseconds - laplacian_x_times - laplacian_y_times - laplacian_z_times -
                            reaction_times - update_times - laplacian_summation_times;

        printf("------------------ %04i ------------------\n", (t+1));
        printf("  Integration time: %12.6f ms\n", milliseconds);
        printf("  Laplacian x:      %12.6f ms\n", laplacian_x_times);
        printf("  Laplacian y:      %12.6f ms\n", laplacian_y_times);
        printf("  Laplacian z:      %12.6f ms\n", laplacian_z_times);
        printf("  Laplacian sum:    %12.6f ms\n", laplacian_summation_times);
        printf("  Reaction:         %12.6f ms\n", reaction_times);
        printf("  Time step:        %12.6f ms\n", update_times);
        printf("  Other:            %12.6f ms\n\n", other_times);
        printf("  Writing results to: %s\n", buffer);
        printf("  Writing data:     %12.6f ms\n\n", elapsed_seconds.count() * 1000);


        // output lowest and highest values
        float min = 100.0;
        float max = 0.0;
        for(unsigned int i=0; i<this->ncells; i++) {
            min = std::min(min, this->a[i]);
        }
        for(unsigned int i=0; i<this->ncells; i++) {
            max = std::max(max, this->a[i]);
        }

        printf("  Min   = %12.6f\n", min);
        printf("  Max   = %12.6f\n", max);
        printf("  Range = %12.6f\n", max - min);

        printf("------------------------------------------\n");
        printf("\n");
    }

    // clean up
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );

    std::cout << std::endl;
    this->cleanup_variables();
}

/**
 * @brief      Initialize all variables
 */
void RD3D::initialize_variables() {
    std::cout << "Loading device variables." << std::endl;

    // build initial concentrations
    std::cout << "Constructing initial concentrations...";
    this->a = new float[this->ncells];
    this->b = new float[this->ncells];

    switch(this->reaction_type) {
        case KINETICS::GRAY_SCOTT:
            this->build_input_central_cube(a, b, 1.0f, 0.0f, 0.5f, 0.25f, 0.05f);
        break;
        case KINETICS::BRUSSELATOR:
            this->build_input_random(a, b, this->c1, this->c2 / this->c1, 0.3);
        break;
        default:
            throw std::runtime_error("Invalid reaction system encountered.");
    }


    std::cout << donestring << std::endl;

    // allocate size on device
    std::cout << "Allocating variables on GPU device... ";
    int bytes = this->ncells * sizeof(float);
    checkCuda( hipMalloc((void**)&this->d_a, bytes) );
    checkCuda( hipMalloc((void**)&this->d_b, bytes) );
    checkCuda( hipMalloc((void**)&this->d_dx2, bytes) );
    checkCuda( hipMalloc((void**)&this->d_dy2, bytes) );
    checkCuda( hipMalloc((void**)&this->d_dz2, bytes) );
    checkCuda( hipMalloc((void**)&this->d_ra, bytes) );
    checkCuda( hipMalloc((void**)&this->d_rb, bytes) );
    checkCuda( hipMalloc((void**)&this->d_da, bytes) );
    checkCuda( hipMalloc((void**)&this->d_db, bytes) );
    std::cout << donestring << std::endl;

    // copy data to device
    std::cout << "Copying data to GPU device...         ";
    checkCuda( hipMemcpy(this->d_a, this->a, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(this->d_b, this->b, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemset(this->d_dx2, 0, bytes) );
    checkCuda( hipMemset(this->d_dy2, 0, bytes) );
    checkCuda( hipMemset(this->d_dz2, 0, bytes) );
    checkCuda( hipMemset(this->d_ra, 0, bytes) );
    checkCuda( hipMemset(this->d_rb, 0, bytes) );
    checkCuda( hipMemset(this->d_da, 0, bytes) );
    checkCuda( hipMemset(this->d_db, 0, bytes) );
    std::cout << donestring << std::endl;

    // set constants
    std::cout << "Setting constant variables on GPU...  ";
    float _diffcon_a = this->Da / (this->dx * this->dx);
    float _diffcon_b = this->Db / (this->dx * this->dx);
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_diffcon_a), &_diffcon_a, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_diffcon_b), &_diffcon_b, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &this->dt, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_mx), &this->mx, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_my), &this->my, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_mz), &this->mz, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_pencils), &this->pencils, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_ncells), &this->ncells, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_c1), &this->c1, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_c2), &this->c2, sizeof(float)) );
    std::cout << donestring << std::endl;

    std::cout << "All ready for time-integration." << std::endl << std::endl;
}

/**
 * @brief      Clean-up all variables
 */
void RD3D::cleanup_variables() {
    std::cout << "Cleaning Integration variables...     ";
    checkCuda( hipFree(this->d_a) );
    checkCuda( hipFree(this->d_b) );
    checkCuda( hipFree(this->d_ra) );
    checkCuda( hipFree(this->d_rb) );
    checkCuda( hipFree(this->d_da) );
    checkCuda( hipFree(this->d_db) );
    checkCuda( hipFree(this->d_dx2) );
    checkCuda( hipFree(this->d_dy2) );
    checkCuda( hipFree(this->d_dz2) );

    delete [] this->a;
    delete [] this->b;

    std::cout << donestring << std::endl;
    std::cout << std::endl;
}

/**
 * @brief      Build random input
 *
 * @param      a      Concentration of a
 * @param      b      Concentration of b
 * @param[in]  a0     initial value a
 * @param[in]  b0     initial value b
 * @param[in]  ca     central concentration for a
 * @param[in]  cb     central concentration for b
 * @param[in]  delta  perturbation strength
 */
void RD3D::build_input_central_cube(float* a, float* b, float a0, float b0, float ca, float cb, float delta) {
    // initialize with random data
    for(unsigned int i=0; i < this->ncells; i++) {
        a[i] = a0 + uniform_dist() * delta;
        b[i] = b0 + uniform_dist() * delta;
    }

    const unsigned int cbsz = 5;
    for(unsigned int z=this->mz/2-cbsz; z<this->mz/2+cbsz; z++) {
        for(unsigned int y=this->my/2-cbsz; y<this->my/2+cbsz; y++) {
            for(unsigned int x=this->mx/2-cbsz; x<this->mx/2+cbsz; x++) {
                a[z * this->mx * this->my + y * this->mx + x] = ca  + uniform_dist() * delta;
                b[z * this->mx * this->my + y * this->mx + x] = cb  + uniform_dist() * delta;
            }
        }
    }
}

/**
 * @brief      Build random input
 *
 * @param      a      Concentration of a
 * @param      b      Concentration of b
 * @param[in]  ca     central concentration for a
 * @param[in]  cb     central concentration for b
 * @param[in]  delta  perturbation strength
 */
void RD3D::build_input_random(float* a, float* b, float ca, float cb, float delta) {
    // initialize with random data
    for(unsigned int i=0; i < this->ncells; i++) {
        a[i] = ca + uniform_dist() * delta;
        b[i] = cb + uniform_dist() * delta;
    }
}

/**
 * @brief      Write 3D concentration profile as binary file
 *
 * @param[in]  filename  The filename
 * @param[in]  vals      Concentration data
 */
void RD3D::write_binary(const std::string filename, const float *vals) {
    std::ofstream out(filename, std::ios::binary);

    // write data size
    uint16_t dim = 0;

    // write size
    dim = this->mx;
    out.write((char*)&dim, sizeof(uint16_t));
    dim = this->my;
    out.write((char*)&dim, sizeof(uint16_t));
    dim = this->mz;
    out.write((char*)&dim, sizeof(uint16_t));
    dim = sizeof(float);
    out.write((char*)&dim, sizeof(uint16_t));

    // write values
    out.write((const char*)vals, sizeof(float) * this->ncells);

    out.close();
}
