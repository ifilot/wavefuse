 /*************************************************************************
 *   This file is part of Wavefuse                                        *
 *   https://gitlab.tue.nl/ifilot/Wavefuse                                *
 *                                                                        *
 *   Author: Ivo Filot <i.a.w.filot@tue.nl>                               *
 *                                                                        *
 *   Wavefuse is free software: you can redistribute it and/or modify     *
 *   it under the terms of the GNU General Public License as published    *
 *   by the Free Software Foundation, either version 3 of the License,    *
 *   or (at your option) any later version.                               *
 *                                                                        *
 *   Wavefuse is distributed in the hope that it will be useful,          *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty          *
 *   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.              *
 *   See the GNU General Public License for more details.                 *
 *                                                                        *
 *   You should have received a copy of the GNU General Public License    *
 *   along with this program.  If not, see http://www.gnu.org/licenses/.  *
 *                                                                        *
 **************************************************************************/

#include "card_manager.h"

CardManager::CardManager() {}

void CardManager::probe_cards() {
    int nDevices;

    std::cout << "GPU DEVICE INFORMATION" << std::endl << std::endl;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "    Device name: " << prop.name << std::endl;
        std::cout << "    Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
        std::cout << "    Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "    Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl << std::endl;
    }

    std::cout << "----------------------------------------------------------" << std::endl;
}
