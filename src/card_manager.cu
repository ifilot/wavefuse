 /*************************************************************************
 *   This file is part of Wavefuse                                        *
 *   https://gitlab.tue.nl/ifilot/Wavefuse                                *
 *                                                                        *
 *   Author: Ivo Filot <i.a.w.filot@tue.nl>                               *
 *                                                                        *
 *   Wavefuse is free software: you can redistribute it and/or modify     *
 *   it under the terms of the GNU General Public License as published    *
 *   by the Free Software Foundation, either version 3 of the License,    *
 *   or (at your option) any later version.                               *
 *                                                                        *
 *   Wavefuse is distributed in the hope that it will be useful,          *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty          *
 *   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.              *
 *   See the GNU General Public License for more details.                 *
 *                                                                        *
 *   You should have received a copy of the GNU General Public License    *
 *   along with this program.  If not, see http://www.gnu.org/licenses/.  *
 *                                                                        *
 **************************************************************************/

#include "card_manager.h"

/**
 * @brief      Constructs the object.
 */
CardManager::CardManager() {

}

/**
 * @brief      Probe the available GPU units
 */
void CardManager::probe_cards() {
    int nDevices;

    std::cout << "GPU DEVICE INFORMATION" << std::endl << std::endl;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "    Device name: " << prop.name << std::endl;
        // std::cout << "    Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
        // std::cout << "    Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "    Number of cores: " << this->get_number_cores((void*)&prop) << std::endl;
        std::cout << "    Number of MP units: " << prop.multiProcessorCount << std::endl;
        std::cout << "    Maximum threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "    Maximum global memory: " << (prop.totalGlobalMem)/(double)(1024*1024) << " MB" << std::endl;
        // std::cout << "    Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl << std::endl;
    }

    std::cout << "----------------------------------------------------------" << std::endl;
}

/**
 * @brief      Get the total number of cores on the GPU
 *
 * @param      devProp  Card properties
 *
 * @return     The number cores.
 */
int CardManager::get_number_cores(void* prop) {
    // below is a nasty hack to convert a void pointer to a hipDeviceProp_t type which
    // is unknown by the C++ compiler when handling the .h file
    hipDeviceProp_t devProp = *(hipDeviceProp_t*)prop;

    int cores = 0;
    int mp = devProp.multiProcessorCount;

    switch (devProp.major){
        case 2: // Fermi
            if (devProp.minor == 1) {
                cores = mp * 48;
            } else {
                cores = mp * 32;
            }
        break;
        case 3: // Kepler
            cores = mp * 192;
        break;
        case 5: // Maxwell
            cores = mp * 128;
        break;
        case 6: // Pascal
            if (devProp.minor == 1) {
                cores = mp * 128;
            } else if (devProp.minor == 0) {
                cores = mp * 64;
            } else {
                std::cerr << "Unknown device type" << std::endl;
            }
        break;
        case 7: // Volta
            if (devProp.minor == 0) {
                cores = mp * 64;
            } else {
                std::cerr << "Unknown device type" << std::endl;
            }
        break;
        default:
            std::cerr << "Unknown device type" << std::endl;
        break;
    }

    return cores;
}
